
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <iomanip>
#include <omp.h>
//enable OMP
//#define OPENMP_ENABLE

#define MAX(x, y) ((x>y) ? x : y)
// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
	if (stat != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
	}
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
	}
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
	if (stat != HIPRAND_STATUS_SUCCESS) {
		fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
	}
}


double cal_tflops(int m, int n, int k, double msec)
{
    double flops = 2. * m * n * k;
    double tflops = (1E-12*flops) / (1E-3*msec);
    return tflops;
}

 

__global__ void assignFloatValue (float *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}

__global__ void assignHalfValue (half *out, int n, float value) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < n) {
		out[idx] = value;
	}
}
void correctnessCheck(int m, int n, int k, float *host, float value){
        for (int i = 0; i < m * n; i++) {      
            float val = host[i];
            if ( val != k * value * value) {
                std::cout << "ERROR value = " << val<< std::endl;
            }
        }
}

void printTime(float cublasTime, int m, int n, int k, float &s_max_tflops, int &s_max_m_n, int &s_max_k ){
        float tflops = cal_tflops(m, n, k, cublasTime);
        if (tflops > s_max_tflops){
            s_max_tflops = tflops;
	    s_max_m_n = m;
            s_max_k = k;
        }
        std::cout << std::setw(7) << m << ",";
        std::cout << std::setw(7) << n << ",";
        std::cout << std::setw(7) << k << ",";
        std::cout << std::setw(15) << std::setprecision(4) << cublasTime << ",";
        std::cout << std::setw(15) << std::setprecision(4) << tflops << "," << std::endl;
}
// find mnk
void findMaxMNK(int argc, char* argv[], int* mnk ){
    int m,n,k;
    std::string precision="NULL";

    
    // precision = INT8_TENSOR
    // precision = FP16_TENSOR
    // precision = FP16_32_TENSOR
    // precision = FP32_CUDA
    // precision = FP16_CUDA
    if (argc == 2 || argc == 3) {
        precision = argv[1];
    }
    
    float s_max_tflops = 0;
    int s_max_m_n = 0;
    int s_max_k = 0;
    
    if (precision == "INT8_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore INT8(INT32 accumulation) Time and TOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TOPS";
    std::cout << std::endl;
    
    // for tensorcore test TODO: to verify the int8 with int8 accumulation
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 5120; k+=1024) {
  
        int8_t *a_;
        int8_t *b_;
        int *c_cublas;
        int *c_host_cublas;
        //const int  value = 1;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_, m * k * sizeof(int8_t)));
        cudaErrCheck(hipMalloc((void**)&b_, k * m * sizeof(int8_t)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(int)));
        c_host_cublas = (int*)malloc(m * n * sizeof(int));

        //TODO curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        //assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        //assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        int alpha = 1;
        int beta = 0;
        int numRepeats = 1;
        // Warp up not really needed here as many params will be tested
        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_, HIP_R_8I, m,
                    b_, HIP_R_8I, n,
                    &beta, 
                    c_cublas, HIP_R_32I, m,
                    HIP_R_32I, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_));
        cudaErrCheck(hipFree(b_));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    

    if (precision == "FP16_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore FP16(FP16 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    s_max_m_n = 0;
    s_max_k = 0;
    // for tensorcore test
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 5120; k+=1024) {
        half *a_fp16;
        half *b_fp16;
        half *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 1;

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, n,
                    &beta, 
                    c_cublas, HIP_R_16F, m,
                    HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
        //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    
 
    if (precision == "FP16_32_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore FP16(FP32 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for tensorcore test
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 5120; k+=1024) {
        half *a_fp16;
        half *b_fp16;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
        hipblasHandle_t cublasHandle;
        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // Use tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 1;
        // Warp up not really needed
        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                    m, n, k, 
                    &alpha,
                    a_fp16, HIP_R_16F, m,
                    b_fp16, HIP_R_16F, n,
                    &beta, 
                    c_cublas, HIP_R_32F, m,
                    HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        cublasTime /= numRepeats;
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
	
    }

    if (precision == "FP32_CUDA" || precision == "NULL") {
    std::cout << "[CUDA core FP32 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for float test
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 5120; k+=1024) {
        float *a_fp32;
        float *b_fp32;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp32, m * k * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&b_fp32, k * n * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignFloatValue <<< (m * k + 255) / 256, 256 >>> (a_fp32, m*k, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (b_fp32, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 1;
        
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                &alpha,
                a_fp32, m,
                b_fp32, n,
                &beta,
                c_cublas, m);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime = 0.0f;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp32));
        cudaErrCheck(hipFree(b_fp32));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
    

    if (precision == "FP16_CUDA" || precision == "NULL") {
    std::cout << "[CUDA core FP16 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for float test
    for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    for(k=1024; k <= 5120; k+=1024) {
        half *a_fp16;
        half *b_fp16;
        half *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;
   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
        assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        half alpha = 1.0f;
        half beta = 0.0f;
        int numRepeats = 1;

        // Now using cuBLAS
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasHgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                &alpha,
                a_fp16, m,
                b_fp16, n,
                &beta,
                c_cublas, m);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // TODO: Correctness check
        //cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
       //correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp16));
        cudaErrCheck(hipFree(b_fp16));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    }}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
    }
	mnk[0]=s_max_m_n;
	mnk[1]=s_max_k;
}

//Stress
void stress(int argc, char* argv[], int* mnk ){
    int m,n,k;
	
	//setup the mnk
	m=mnk[0];
	n=mnk[0];
	k=mnk[1];
    std::string precision="NULL";
	
#ifdef OPENMP_ENABLE

	int num_gpus;
    cudaGetDeviceCount(&num_gpus);
	printf("Num GPU->%d \n",num_gpus);
	
	
	 if(num_gpus < 1)
    {
	    printf("no CUDA capable devices were detected\n");
        exit(1);
    }
//OMP starting...	
#pragma omp parallel num_threads(num_gpus)
{  
	unsigned int cpu_thread_id = omp_get_thread_num();
	int gpu_id = -1;
	cudaSetDevice(cpu_thread_id % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
	cudaGetDevice(&gpu_id);
	printf("GPU->%d \n",gpu_id);
	
#endif
    
    // precision = INT8_TENSOR
    // precision = FP16_TENSOR
    // precision = FP16_32_TENSOR
    // precision = FP32_CUDA
    // precision = FP16_CUDA
    if (argc == 2 || argc == 3) {
        precision = argv[1];
    }
    
    float s_max_tflops = 0;
    int s_max_m_n = 0;
    int s_max_k = 0;
    
    if (precision == "INT8_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore INT8(INT32 accumulation) Time and TOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TOPS";
    std::cout << std::endl;
    
		// for tensorcore test TODO: to verify the int8 with int8 accumulation
		//for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
		//for(k=1024; k <= 5120; k+=1024) {
		while(true){
	  
			int8_t *a_;
			int8_t *b_;
			int *c_cublas;
			int *c_host_cublas;
			//const int  value = 1;

	   
			hipblasHandle_t cublasHandle;

			hipEvent_t startcublas;
			hipEvent_t stopcublas;

			cudaErrCheck(hipEventCreate(&startcublas));
			cudaErrCheck(hipEventCreate(&stopcublas));
			cublasErrCheck(hipblasCreate(&cublasHandle));
			// Use tensor cores
			cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

			cudaErrCheck(hipMalloc((void**)&a_, m * k * sizeof(int8_t)));
			cudaErrCheck(hipMalloc((void**)&b_, k * m * sizeof(int8_t)));
			cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(int)));
			c_host_cublas = (int*)malloc(m * n * sizeof(int));

			//TODO curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
			//assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
			//assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
			//assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

			int alpha = 1;
			int beta = 0;
			int numRepeats = 1;
			// Warp up not really needed here as many params will be tested
			// Now using cuBLAS
			cudaErrCheck(hipEventRecord(startcublas));
			for (int iteration = 0; iteration < numRepeats; ++iteration) {
			cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
						m, n, k, 
						&alpha,
						a_, HIP_R_8I, m,
						b_, HIP_R_8I, n,
						&beta, 
						c_cublas, HIP_R_32I, m,
						HIP_R_32I, CUBLAS_GEMM_DFALT_TENSOR_OP));
			}
			cudaErrCheck(hipEventRecord(stopcublas));
			cudaErrCheck(hipEventSynchronize(stopcublas));
			// TODO: Correctness check
			//cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
		   //correctnessCheck(m, n, k, c_host_cublas, value);
			// Check time
			float cublasTime;	
			cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
			cublasTime /= numRepeats;
			printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
			
			cudaErrCheck(hipEventDestroy(startcublas));             
			cudaErrCheck(hipEventDestroy(stopcublas));
			cudaErrCheck(hipFree(a_));
			cudaErrCheck(hipFree(b_));
			cudaErrCheck(hipFree(c_cublas));
			free(c_host_cublas);
		}  //}}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
    cudaErrCheck(hipDeviceReset());
	#ifdef OPENMP_ENABLE
	#pragma omp barrier
	cudaDeviceSynchronize();
	#endif 
    }

    if (precision == "FP16_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore FP16(FP16 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    s_max_m_n = 0;
    s_max_k = 0;
    // for tensorcore test
    //for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    //for(k=1024; k <= 5120; k+=1024) {
		while(true){
			half *a_fp16;
			half *b_fp16;
			half *c_cublas;
			float *c_host_cublas;
			const float  value = 1.0f;
	   
			hipblasHandle_t cublasHandle;

			hipEvent_t startcublas;
			hipEvent_t stopcublas;

			cudaErrCheck(hipEventCreate(&startcublas));
			cudaErrCheck(hipEventCreate(&stopcublas));
			cublasErrCheck(hipblasCreate(&cublasHandle));
			// Use tensor cores
			cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

			cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
			c_host_cublas = (float*)malloc(m * n * sizeof(float));

			// curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
			assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
			assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
			assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

			float alpha = 1.0f;
			float beta = 0.0f;
			int numRepeats = 1;

			// Now using cuBLAS
			cudaErrCheck(hipEventRecord(startcublas));
			for (int iteration = 0; iteration < numRepeats; ++iteration) {
			hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
						m, n, k, 
						&alpha,
						a_fp16, HIP_R_16F, m,
						b_fp16, HIP_R_16F, n,
						&beta, 
						c_cublas, HIP_R_16F, m,
						HIP_R_16F, CUBLAS_GEMM_DFALT_TENSOR_OP);
			}
			cudaErrCheck(hipEventRecord(stopcublas));
			cudaErrCheck(hipEventSynchronize(stopcublas));
			// TODO: Correctness check
			//cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
			//correctnessCheck(m, n, k, c_host_cublas, value);
			// Check time
			float cublasTime;	
			cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
			printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
			
			cudaErrCheck(hipEventDestroy(startcublas));             
			cudaErrCheck(hipEventDestroy(stopcublas));
			cudaErrCheck(hipFree(a_fp16));
			cudaErrCheck(hipFree(b_fp16));
			cudaErrCheck(hipFree(c_cublas));
			free(c_host_cublas);
		}//}}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
	#ifdef OPENMP_ENABLE
	#pragma omp barrier
	cudaDeviceSynchronize();
	#endif	
    cudaErrCheck(hipDeviceReset());
    }
    
 
    if (precision == "FP16_32_TENSOR" || precision == "NULL") {
    std::cout << "[TensorCore FP16(FP32 accumulation) Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for tensorcore test
    //for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    //for(k=1024; k <= 5120; k+=1024) {
		while(true){
			half *a_fp16;
			half *b_fp16;
			float *c_cublas;
			float *c_host_cublas;
			const float  value = 1.0f;
			hipblasHandle_t cublasHandle;
			hipEvent_t startcublas;
			hipEvent_t stopcublas;

			cudaErrCheck(hipEventCreate(&startcublas));
			cudaErrCheck(hipEventCreate(&stopcublas));
			cublasErrCheck(hipblasCreate(&cublasHandle));
			// Use tensor cores
			cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

			cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
			c_host_cublas = (float*)malloc(m * n * sizeof(float));

			// curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
			assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
			assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
			assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

			float alpha = 1.0f;
			float beta = 0.0f;
			int numRepeats = 1;
			// Warp up not really needed
			// Now using cuBLAS
			cudaErrCheck(hipEventRecord(startcublas));
			for (int iteration = 0; iteration < numRepeats; ++iteration) {
			cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
						m, n, k, 
						&alpha,
						a_fp16, HIP_R_16F, m,
						b_fp16, HIP_R_16F, n,
						&beta, 
						c_cublas, HIP_R_32F, m,
						HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
			}
			cudaErrCheck(hipEventRecord(stopcublas));
			cudaErrCheck(hipEventSynchronize(stopcublas));
			// Correctness check
			cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
			correctnessCheck(m, n, k, c_host_cublas, value);
			// Check time
			float cublasTime;	
			cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
			cublasTime /= numRepeats;
			printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
			
			cudaErrCheck(hipEventDestroy(startcublas));             
			cudaErrCheck(hipEventDestroy(stopcublas));
			cudaErrCheck(hipFree(a_fp16));
			cudaErrCheck(hipFree(b_fp16));
			cudaErrCheck(hipFree(c_cublas));
			free(c_host_cublas);
		} //}}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
	#ifdef OPENMP_ENABLE
	#pragma omp barrier
	cudaDeviceSynchronize();
	#endif	
    cudaErrCheck(hipDeviceReset());
	
    }

    if (precision == "FP32_CUDA" || precision == "NULL") {
    std::cout << "[CUDA core FP32 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for float test
    //for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    //for(k=1024; k <= 5120; k+=1024) {
	while(true){
        float *a_fp32;
        float *b_fp32;
        float *c_cublas;
        float *c_host_cublas;
        const float  value = 1.0f;

   
        hipblasHandle_t cublasHandle;

        hipEvent_t startcublas;
        hipEvent_t stopcublas;

        cudaErrCheck(hipEventCreate(&startcublas));
        cudaErrCheck(hipEventCreate(&stopcublas));
        cublasErrCheck(hipblasCreate(&cublasHandle));
        // No tensor cores
        cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

        cudaErrCheck(hipMalloc((void**)&a_fp32, m * k * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&b_fp32, k * n * sizeof(float)));
        cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(float)));
        c_host_cublas = (float*)malloc(m * n * sizeof(float));

        // curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
        assignFloatValue <<< (m * k + 255) / 256, 256 >>> (a_fp32, m*k, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (b_fp32, k*n, value);
        assignFloatValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

        float alpha = 1.0f;
        float beta = 0.0f;
        int numRepeats = 1;
        
        cudaErrCheck(hipEventRecord(startcublas));
        for (int iteration = 0; iteration < numRepeats; ++iteration) {
        hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                &alpha,
                a_fp32, m,
                b_fp32, n,
                &beta,
                c_cublas, m);
        }
        cudaErrCheck(hipEventRecord(stopcublas));
        cudaErrCheck(hipEventSynchronize(stopcublas));
        // Correctness check
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), hipMemcpyDeviceToHost));
        correctnessCheck(m, n, k, c_host_cublas, value);
        // Check time
        float cublasTime = 0.0f;	
        cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
        printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
        
        cudaErrCheck(hipEventDestroy(startcublas));             
        cudaErrCheck(hipEventDestroy(stopcublas));
        cudaErrCheck(hipFree(a_fp32));
        cudaErrCheck(hipFree(b_fp32));
        cudaErrCheck(hipFree(c_cublas));
        free(c_host_cublas);
    } //}}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
	#ifdef OPENMP_ENABLE
	#pragma omp barrier
	cudaDeviceSynchronize();
	#endif	
    cudaErrCheck(hipDeviceReset());
    }
    

    if (precision == "FP16_CUDA" || precision == "NULL") {
    std::cout << "[CUDA core FP16 Time and TFLOPS Result]" << std::endl;
    std::cout << std::setw(7) << "m" << std::setw(7) << "n" << std::setw(7) << "k";
    std::cout << std::setw(15) << "Time (msec)" << std::setw(15) << "TFLOPS";
    std::cout << std::endl;
    s_max_tflops = 0;
    // for float test
    //for(m=1024, n = 1024; m <= 25600; m+=1024, n+=1024) {
    //for(k=1024; k <= 5120; k+=1024) {
	while(true){
			half *a_fp16;
			half *b_fp16;
			half *c_cublas;
			float *c_host_cublas;
			const float  value = 1.0f;
	   
			hipblasHandle_t cublasHandle;

			hipEvent_t startcublas;
			hipEvent_t stopcublas;

			cudaErrCheck(hipEventCreate(&startcublas));
			cudaErrCheck(hipEventCreate(&stopcublas));
			cublasErrCheck(hipblasCreate(&cublasHandle));
			// No tensor cores
			cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

			cudaErrCheck(hipMalloc((void**)&a_fp16, m * k * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&b_fp16, k * n * sizeof(half)));
			cudaErrCheck(hipMalloc((void**)&c_cublas, m * n * sizeof(half)));
			c_host_cublas = (float*)malloc(m * n * sizeof(float));

			// curand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
			assignHalfValue <<< (m * k + 255) / 256, 256 >>> (a_fp16, m*k, value);
			assignHalfValue <<< (k * n + 255) / 256, 256 >>> (b_fp16, k*n, value);
			assignHalfValue <<< (k * n + 255) / 256, 256 >>> (c_cublas, m*n, 0.0f);

			half alpha = 1.0f;
			half beta = 0.0f;
			int numRepeats = 1;

			// Now using cuBLAS
			cudaErrCheck(hipEventRecord(startcublas));
			for (int iteration = 0; iteration < numRepeats; ++iteration) {
			hipblasHgemm(cublasHandle,
					HIPBLAS_OP_N,
					HIPBLAS_OP_T,
					m,
					n,
					k,
					&alpha,
					a_fp16, m,
					b_fp16, n,
					&beta,
					c_cublas, m);
			}
			cudaErrCheck(hipEventRecord(stopcublas));
			cudaErrCheck(hipEventSynchronize(stopcublas));
			// TODO: Correctness check
			//cudaErrCheck(cudaMemcpy(c_host_cublas, c_cublas, m * n * sizeof(float), cudaMemcpyDeviceToHost));
		   //correctnessCheck(m, n, k, c_host_cublas, value);
			// Check time
			float cublasTime;	
			cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas)); 
			printTime(cublasTime, m, n, k, s_max_tflops, s_max_m_n, s_max_k);
			
			cudaErrCheck(hipEventDestroy(startcublas));             
			cudaErrCheck(hipEventDestroy(stopcublas));
			cudaErrCheck(hipFree(a_fp16));
			cudaErrCheck(hipFree(b_fp16));
			cudaErrCheck(hipFree(c_cublas));
			free(c_host_cublas);
		} //}}
    std::cout << "[Peak TFLOPS]=" << s_max_tflops << ", m=n="<< s_max_m_n << ", k="<<s_max_k<< std::endl;
	#ifdef OPENMP_ENABLE
	#pragma omp barrier
	cudaDeviceSynchronize();
	#endif
    cudaErrCheck(hipDeviceReset());
    }
	

#ifdef OPENMP_ENABLE
}//end OPENMP_ENABLE
#endif
}


int main(int argc, char* argv[]) {


printf("option 1)Benchmark-> benchmark option 2) Stress in single thread option 3) Stress in multiple thread \n");
int* mnk=(int*)malloc(sizeof(int)*2); //mnk[0]->mn mnk[1]->k

findMaxMNK(argc,argv,mnk);
stress(argc,argv,mnk);

free(mnk);

return 0;
}
